#include "hip/hip_runtime.h"
#include "kmeansTransformGpu.hh"

KmeansTransformGpu::KmeansTransformGpu(const std::string &path, unsigned int nbClusters, unsigned int clusterDim)
        : centroids_(clusterDim, nbClusters),
          cudaCentroids_(clusterDim, nbClusters, nullptr),
          nbClusters_(nbClusters),
          clusterDim_(clusterDim) {
    Matrix<>::readMatrix(path, centroids_);
    unsigned centroidSize = nbClusters * clusterDim * sizeof(Matrix<>::data_t);
    hipMalloc(&(cudaCentroids_.getData()), centroidSize);
    hipMemcpy(cudaCentroids_.getData(), centroids_.getData(), centroidSize, hipMemcpyHostToDevice);
}

KmeansTransformGpu::~KmeansTransformGpu() {
    hipFree(cudaCentroids_.getData());
}

CUDA_DEV float execComputeDistance(const float* clusterCentroid, unsigned centroidWith, const float* feature) {
    float sum = 0;
    for (auto i = 0U; i < centroidWith; ++i) {
        float sub = feature[i] - clusterCentroid[i];
        sum += sub * sub;
    }

    return sqrtf(sum);
}

CUDA_GLOBAL void execTransform(const Matrix<float> cudaFeatures, Matrix<> cudaCentroids, Matrix<unsigned char> cudaLabels) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cudaFeatures.height())
        return;

    // Copy current features in local memory
    float feature[256];
    for (auto i = 0U; i < 256; ++i)
        feature[i] = cudaFeatures[index][i];

    // Get smallest euclidian distance cluster
    float dist = INFINITY;
    unsigned char cluster = 0;
    for (auto j = 0U; j < cudaCentroids.height(); ++j) {
        // float curDist = execComputeDistance(centroids + (j * cudaCentroids.width()), cudaCentroids.width(), cudaFeatures[index]);
        float curDist = execComputeDistance(
            cudaCentroids.getData() + (j * cudaCentroids.width()),
            cudaCentroids.width(),
            feature
        );
        if (curDist < dist) {
            dist = curDist;
            cluster = j;
        }
    }
    cudaLabels[index][0] = cluster;
}

void KmeansTransformGpu::transform(const Matrix<float> &cudaFeatures, std::vector<uchar> &labels) {
    if (cudaFeatures.height() > labels.size())
        throw std::invalid_argument("Invalid label buffer: to small!");

    // Create cuda label buffer
    Matrix<unsigned char> cudaLabels(1, cudaFeatures.height(), nullptr);
    unsigned labelSize = cudaLabels.width() * cudaLabels.height() * sizeof(uchar);
    hipMalloc(&(cudaLabels.getData()), labelSize);

    // Compute Kernel dimensions
    unsigned blockWidth = 256;
    unsigned gridWidth = cudaFeatures.height() / blockWidth;
    if (gridWidth % blockWidth != 0)
        gridWidth += 1;

    // Execute kernel
    unsigned centroidSize = centroids_.width() * centroids_.height() * sizeof(Matrix<>::data_t);
    execTransform<<<gridWidth, blockWidth, centroidSize>>>(cudaFeatures, cudaCentroids_, cudaLabels);

    // Copy result in memory
    hipMemcpy(labels.data(), cudaLabels.getData(), labelSize, hipMemcpyDeviceToHost);
    hipFree(cudaLabels.getData());
}